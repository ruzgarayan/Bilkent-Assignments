#include "hip/hip_runtime.h"
#include <util.h>

//This includes enough information to hold the best solutions so far
//to be later outputted in the form "<k-mer>, <index on reference>, <extension score>"
typedef struct {
    int indexOnRead;
    int indexOnReference;
    int extensionScore;
} Solution;

__global__ void kmerSearchAndExtend(char *d_ref, char *d_reads, Solution *d_solutions, int k,
                                    int referenceLength, int readLength) {
    //printf("%d %d\n", blockIdx.x, threadIdx.x);
    int indexOnRead = threadIdx.x;
    char *readString = d_reads + blockIdx.x * readLength;
    char *kmer = readString + indexOnRead;
    
    int isHit;

    Solution bestKmerHit;
    bestKmerHit.extensionScore = -1;

    //Iterate over all the possible starting indices on the reference string to check a hit
    for (int i = 0; i < referenceLength - k; i++) {
        isHit = 1;
        for (int j = 0; j < k; j++) {
            if (d_ref[i + j] != kmer[j]) {
                isHit = 0;
                break; 
            }
        }

        if (isHit) {
            int extendedReferenceStart = i - indexOnRead;
            int extendedReferenceEnd = extendedReferenceStart + readLength;
            if (extendedReferenceEnd > referenceLength) extendedReferenceEnd = referenceLength;

            int extensionScore = 0;

            //Extend the original read string and count the equal characters for the extension score.
            for (int j = 0; j < readLength && extendedReferenceStart + j < extendedReferenceEnd; j++) {
                if (extendedReferenceStart + j >= 0 && d_ref[extendedReferenceStart + j] == readString[j]) {
                    extensionScore++;
                }
            }

            //Check if this new hit is better than the current best. In that case, change the current best
            if (extensionScore > bestKmerHit.extensionScore 
                || (extensionScore == bestKmerHit.extensionScore && i < bestKmerHit.indexOnReference) ) {
                bestKmerHit.extensionScore = extensionScore;
                bestKmerHit.indexOnReference = i;
                bestKmerHit.indexOnRead = indexOnRead;
            }
            //printf("Hit at reference index %d, extension score is %d\n", i, extensionScore);
        }
    }
    d_solutions[blockIdx.x * blockDim.x + threadIdx.x] = bestKmerHit;
}

int main(int argc, char** argv)
{
    if(argc != 5) {
        printf("Wrong argments usage: ./kmer [REFERENCE_FILE] [READ_FILE] [k] [OUTPUT_FILE]\n" );
        return 1;
    }

    char *referenceStringFile = argv[1];
    char *readStringsFile = argv[2];
    int k = atoi(argv[3]);
    char *outputFile = argv[4];

    char *referenceString = (char*) malloc(MAX_REF_LENGTH * sizeof(char));;

    FILE* file;
    file = fopen(referenceStringFile, "r");
    if (file == 0) {
        printf("Cannot open the reference file %s, exiting the program.\n", referenceStringFile);
        return 1; 
    }

    if (fgets(referenceString, MAX_REF_LENGTH, file) == NULL) { //A single line only
        printf("Problem in file format!\n");
        return 1;
    }
    fclose(file);

    StringList readStringList;

    initStringList(&readStringList, 10);  // initially 5 elements
    int successReadFile = read_file(readStringsFile, &readStringList);

    if (successReadFile < 0) {
        printf("Cannot read the reads file %s, exiting the program!\n", readStringsFile);
        return 1;
    }

    int numReadStrings = readStringList.used;
    int referenceLength = strlen(referenceString);
    int readLength = strlen(readStringList.array[0]);


    dim3 DimGrid(numReadStrings); // 1 block for every read string
    int threadsPerBlock = readLength - k + 1;
    dim3 DimBlock(threadsPerBlock); // 1 thread for every kmer of the read string of that block

    //Flatten the read strings into a single string
    char *flattenedReads = (char*) malloc(readLength * numReadStrings * sizeof(char));;
    for (int i = 0; i < numReadStrings; i++) {
        memcpy(flattenedReads + i * readLength, readStringList.array[i], readLength);
    }

    //Declare pointers to device memory
    char *d_ref;
    char *d_reads;
    Solution *d_solutions;

    hipMalloc((void **) &d_ref, referenceLength * sizeof(char));
    hipMalloc((void **) &d_reads, readLength * numReadStrings * sizeof(char));
    hipMalloc((void **) &d_solutions, threadsPerBlock * numReadStrings * sizeof(Solution));

    hipMemcpy(d_ref, referenceString, referenceLength * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_reads, flattenedReads, readLength * numReadStrings * sizeof(char), hipMemcpyHostToDevice);

    kmerSearchAndExtend<<<DimGrid, DimBlock>>>(d_ref, d_reads, d_solutions, k, referenceLength, readLength);
    hipDeviceSynchronize();

    Solution *h_solutions = (Solution *) malloc(threadsPerBlock * numReadStrings * sizeof(Solution));
    hipMemcpy(h_solutions, d_solutions, threadsPerBlock * numReadStrings * sizeof(Solution), hipMemcpyDeviceToHost);

    Solution *allBestHits = (Solution *) malloc(numReadStrings * sizeof(Solution));

    for (int i = 0; i < numReadStrings; i++) {
        Solution bestSoFar;
        bestSoFar.extensionScore = -1;
        for (int j = 0; j < threadsPerBlock; j++) {
            Solution currSolution = h_solutions[i * threadsPerBlock + j];
            
            if (currSolution.extensionScore > bestSoFar.extensionScore 
                || (currSolution.extensionScore == bestSoFar.extensionScore && currSolution.indexOnReference < bestSoFar.indexOnReference) ) {
                bestSoFar.extensionScore = currSolution.extensionScore;
                bestSoFar.indexOnReference = currSolution.indexOnReference;
                bestSoFar.indexOnRead = currSolution.indexOnRead;
            }
        }

        allBestHits[i] = bestSoFar;
    }

    file = fopen(outputFile, "w");
    if (file == 0) {
        printf("Cannot open the output file %s, exiting the program.\n", outputFile);
        return 1; 
    }

    char kmer[MAX_READ_LENGTH];
    for (int i = 0; i < numReadStrings; i++) {
        char *currReadString = readStringList.array[i];
        Solution bestHit = allBestHits[i];
        if (bestHit.extensionScore < 0) {
            fprintf(file, "No hit has been found for read string %d\n", i + 1);
            continue;
        }

        for (int j = 0; j < k; j++)
            kmer[j] = currReadString[j + bestHit.indexOnRead];
        kmer[k] = '\0';

        fprintf(file, "%s, %d, %d\n", kmer, bestHit.indexOnReference, bestHit.extensionScore);
    }

    fclose(file);

    free(allBestHits);
    free(h_solutions);
    freeStringList(&readStringList);
    free(flattenedReads);
    free(referenceString);
    return 0;
}